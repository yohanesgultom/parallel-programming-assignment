// author: yohanes.gultom@gmail.com
// Source adapted from: https://raw.githubusercontent.com/sol-prog/cuda_cublas_curand_thrust/master/mmul_1.cu
// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <time.h>

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            printf("%.2f ", A[j * nr_rows_A + i]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(int argc, char** argv) {
	if (argc < 5) {
		printf("unsufficient arguments\n");
		return EXIT_FAILURE;
	}


	// Allocate 3 arrays on CPU
	int nr_rows_A = atoi(argv[1]);
	int nr_cols_A = atoi(argv[2]);
	int nr_rows_B = nr_cols_A;
	int nr_cols_B = atoi(argv[3]);
	int nr_rows_C = nr_cols_A;
	int nr_cols_C = nr_rows_B;
	int reps = atoi(argv[4]);

	// // for simplicity we are going to use square arrays
	// nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 3;

	float *h_A, *h_B, *h_C;
	float *d_A, *d_B, *d_C;
	double total_time = 0.0;
	int i = 0;
	for (i = 0; i < reps; i++) {
		double exec_time = ((double) clock()) * -1;

		h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
		h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
		h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

		// Allocate 3 arrays on GPU
		hipMalloc(&d_A,nr_rows_A * nr_cols_A * sizeof(float));
		hipMalloc(&d_B,nr_rows_B * nr_cols_B * sizeof(float));
		hipMalloc(&d_C,nr_rows_C * nr_cols_C * sizeof(float));

		// If you already have useful values in A and B you can copy them in GPU:
		// cudaMemcpy(d_A,h_A,nr_rows_A * nr_cols_A * sizeof(float),cudaMemcpyHostToDevice);
		// cudaMemcpy(d_B,h_B,nr_rows_B * nr_cols_B * sizeof(float),cudaMemcpyHostToDevice);

		// Fill the arrays A and B on GPU with random numbers
		GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
		GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

		// Optionally we can copy the data back on CPU and print the arrays
		hipMemcpy(h_A,d_A,nr_rows_A * nr_cols_A * sizeof(float),hipMemcpyDeviceToHost);
		hipMemcpy(h_B,d_B,nr_rows_B * nr_cols_B * sizeof(float),hipMemcpyDeviceToHost);
		// printf("A:\n");
		// print_matrix(h_A, nr_rows_A, nr_cols_A);
		// printf("B:\n");
		// print_matrix(h_B, nr_rows_B, nr_cols_B);

		// Multiply A and B on GPU
		gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

		// Copy (and print) the result on host memory
		hipMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),hipMemcpyDeviceToHost);
		// printf("C:\n");
		// print_matrix(h_C, nr_rows_C, nr_cols_C);

		//Free GPU memory
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);

		// Free CPU memory
		free(h_A);
		free(h_B);
		free(h_C);

		total_time = total_time + (exec_time + ((double)clock())) / CLOCKS_PER_SEC;
		// printf("%d: %.6f\n", i, ((exec_time + ((double)clock())) / CLOCKS_PER_SEC));
	}
	printf("%d\t%d\t%d\t%d\t%.6f\n", nr_rows_A, nr_cols_A, nr_cols_B, reps, (total_time/reps));
	return EXIT_SUCCESS;
}
