// incrementArray.cu

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

void incrementArrayOnHost(float *a, int N)
{
    int i;
    for (i=0; i < N; i++) a[i] = a[i]+1.f;
}

__global__ void incrementArrayOnDevice(float *a, int N)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<N) a[idx] = a[idx]+1.f;
}

int main(void)
{
    float *a_h, *b_h; // pointers to host memory
    float *a_d; // pointer to device memory
    int i, N = 10;
    size_t size = N*sizeof(float);

    // allocate arrays on host
    a_h = (float *)malloc(size);
    b_h = (float *)malloc(size);
    // allocate array on device
    hipMalloc((void **) &a_d, size);
    // initialization of host data
    for (i=0; i<N; i++) a_h[i] = (float)i;
    // copy data from host to device
    hipMemcpy(a_d, a_h, sizeof(float)*N, hipMemcpyHostToDevice);
    // do calculation on host
    incrementArrayOnHost(a_h, N);
    // do calculation on device:
    // Part 1 of 2. Compute execution configuration
    int blockSize = 4;
    int nBlocks = N/blockSize + (N%blockSize == 0?0:1);
    // Part 2 of 2. Call incrementArrayOnDevice kernel
    incrementArrayOnDevice <<< nBlocks, blockSize >>> (a_d, N);
    // Retrieve result from device and store in b_h
    hipMemcpy(b_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    // check results
    for (i=0; i<N; i++) assert(a_h[i] == b_h[i]);
    // cleanup
    free(a_h); free(b_h); hipFree(a_d);
}
